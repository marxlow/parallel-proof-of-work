#include "hip/hip_runtime.h"
#include <iostream>
#include <bitset>
#include <string>
#include <stdint.h>
#include <stdio.h>
#include <fstream>
#include <ctime>
#include <cmath>

#include "hash.h" // hash.h must be in the current directory
using namespace std;

__device__ uint8_t device_prepend_byte_array[44];
__device__ unsigned long long n;
// These 2 variables will only be set once an answer is found by a thread.
// Copy these values back into host.
__device__ long long nonce_answer;
__device__ uint8_t digest_answer[32];
__device__ bool found_res;


// Helper functions
const char* hex_char_to_bin(char c) {
    switch(toupper(c)) {
        case '0': return "0000";
        case '1': return "0001";
        case '2': return "0010";
        case '3': return "0011";
        case '4': return "0100";
        case '5': return "0101";
        case '6': return "0110";
        case '7': return "0111";
        case '8': return "1000";
        case '9': return "1001";
        case 'A': return "1010";
        case 'B': return "1011";
        case 'C': return "1100";
        case 'D': return "1101";
        case 'E': return "1110";
        case 'F': return "1111";
    }
    return "0000";
}

std::string hex_str_to_bin_str(const std::string& hex)
{
    // TODO use a loop from <algorithm> or smth
    std::string bin;
    for(unsigned i = 0; i != hex.length(); ++i)
       bin += hex_char_to_bin(hex[i]);
    return bin;
}

// Parallel code/functions
__global__ void find_nonce() {
    printf("Thread ID: %d\n", threadIdx.x);
    printf("The value here: %d\n", device_prepend_byte_array[0]);
    found_res = false;
    // Step 1: Calculate the value of X which is 416 bit in length and defined as:
    // (416, 384] t --> Unix timestamp (seconds since UNIX epoch), unsigned 32-bit number.
    // [383, 128] previous_digest --> 256 bits given as input
    // [127, 64] id -->  NUSNET ID "E0003049" in char representation
    // [63, 0] nonce --> unsigned 64-bit number. Can be in the range of [2^64 - 1, 0]. This is what we have to find
    long long nonce = blockDim.x * blockIdx.x + threadIdx.x;
    uint8_t nonce_bytes[8];
    int mask = 0xFF;
    // Reversed due to big endianness
    nonce_bytes[7] = (int)(nonce & mask);
    nonce_bytes[6] = (int)((nonce>>8) & mask);
    nonce_bytes[5] = (int)((nonce>>16) & mask);
    nonce_bytes[4] = (int)((nonce>>24) & mask);
    nonce_bytes[3] = (int)((nonce>>32) & mask);
    nonce_bytes[2] = (int)((nonce>>40) & mask);
    nonce_bytes[1] = (int)((nonce>>48) & mask);
    nonce_bytes[0] = (int)((nonce>>56) & mask);
    uint8_t x[52];
    for (int i = 0 ; i < 44; i++) {
        x[i] = device_prepend_byte_array[i];
    }
    for (int i = 44; i < 52; i++) {
        x[i] = nonce_bytes[i - 44];
    }
    // Step 2: Hash SHA256(X)
    //__device__ void hash.sha256(uint8_t hash[32], const uint8_t * input, size_t len);
    uint8_t hash_res[32]; 
    sha256(hash_res, x, sizeof(x));

    // Step 3: Get first 64-bits of the digest SHA256(X)
    unsigned long long digest = 0; 
    unsigned long long hash_res_long = (unsigned long long)(hash_res[0]);
    digest += hash_res_long<<56;
    hash_res_long = (unsigned long long)(hash_res[1]);
    digest += hash_res_long<<48;
    hash_res_long = (unsigned long long)(hash_res[2]);
    digest += hash_res_long<<40;
    hash_res_long = (unsigned long long)(hash_res[3]);
    digest += hash_res_long<<32;
    hash_res_long = (unsigned long long)(hash_res[4]);
    digest += hash_res_long<<24;  
    hash_res_long = (unsigned long long)(hash_res[5]);
    digest += hash_res_long<<16;  
    hash_res_long = (unsigned long long)(hash_res[6]);
    digest += hash_res_long<<8;  
    hash_res_long = (unsigned long long)(hash_res[7]);
    digest += hash_res_long;  
    printf("digest = %lu\n ", digest);
    
    // Step 4: Compare with "n" to see if it can be accepted
    // TODO(lowjiansheng): pass these values back to host
    if (digest < n) {
        found_res = true;
        nonce_answer = nonce;
        digest_answer = hash_res;
    }
}

int main(int argc, char **argv) {
    
    // Initialize x with 416 bits of 0s
    std::bitset<416> x;
    
    printf("~~~~~~~~~~ Calculating proof of work ~~~~~~~~~~ \n");
    printf("> Reading input file.... '%s'\n", argv[1]);
    std::ifstream file(argv[1]);
    if (file.is_open()) {
        // Read odd lines (1, 3, 5...) as digest
        // Read even lines(2, 4, 6...) as n in decimal form
        std::string previous_digest;
        while (getline(file, previous_digest)) {
            std::string n_decimal;
            getline(file, n_decimal);
            
            printf("> Pre-processing digest & decimal\n");
            // Convert Unix timestamp of time now (seconds since UNIX epoch) to  bit array
            std::time_t time_now = std::time(0); // unsigned(?) 32-bit integer
            std::bitset<32> t_bit(time_now);

            // Convert previous_digest in hex format(Each hex value corresponds to 4-bits) to bit array
            std::string previous_digest_binary_string = hex_str_to_bin_str(previous_digest);
            std::bitset<256> previous_digest_bit(string(previous_digest_binary_string.c_str()));
            
            // Convert NUSNET ID to bit array
            std::string nus_net_id = "E0003049";
            std::string nus_net_id_binary;
            for(int i = 0; i < nus_net_id.length(); i ++) {
                std::bitset<8> char_in_bit(nus_net_id[i]);
                nus_net_id_binary += char_in_bit.to_string();
            }
            std::bitset<64> id_bit(string(nus_net_id_binary.c_str()));
            
            // Concatenate "[t_bit] [previous_digest_bit] [id_bit]"
            std::bitset<352>prepend_bit(
                t_bit.to_string() + 
                previous_digest_bit.to_string() +
                id_bit.to_string()
            );

            // Convert bitset to a uint_8t array
            uint8_t prepend_array[44];
            for (int i = 0; i < prepend_bit.size() - 7; i += 8) {
                int byte_value = 
                    prepend_bit.test(i) * pow(2, 7)+
                    prepend_bit.test(i+1) * pow(2, 6)+
                    prepend_bit.test(i+2) * pow(2, 5)+
                    prepend_bit.test(i+3) * pow(2, 4)+
                    prepend_bit.test(i+4) * pow(2, 3)+
                    prepend_bit.test(i+5) * pow(2, 2)+
                    prepend_bit.test(i+6) * pow(2, 1)+
                    prepend_bit.test(i+7);
                uint8_t byte_value_uint8 = byte_value;
                int index = i / 8;
                prepend_array[index] = byte_value_uint8;

            }
            printf("> Pre-processing done\n");
            hipError_t rc = hipMemcpyToSymbol(HIP_SYMBOL(device_prepend_byte_array), &prepend_array, sizeof(device_prepend_byte_array));
            if (rc != hipSuccess) {
                printf("Could not copy to device. Reason %s\n", hipGetErrorString(rc));
            }
            printf("> Passing 352-bitset to threads to find nonce\n");
            // TODO(lowjiansheng): Find some way to check if a thread has found the answer. Once found terminate and return.
            find_nonce<<<1, 2>>>(); // (num_thread_blocks, num_threads/block)
            hipDeviceSynchronize(); // Waits for all CUDA threads to complete.
        }
        file.close();
    }
    printf("~~~~~~~~~~ Done ~~~~~~~~~~ \n");
    return 0;
}